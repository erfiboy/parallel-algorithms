#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "bmm.h"
#include "math.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// TILEX and TILEY are used to set the number of threads in a CUDA block 
#define TILEX 32
#define TILEY 16

// you may define other parameters here!
#define array2d(array, i, j)	array[i*n + j]

#define factor 4

#if TILEX > TILEY
	#define common_dim_of_matrixes (factor * TILEX)
#else
	#define common_dim_of_matrixes (factor * TILEY)
#endif
// you may define other functions here!


// 
struct SubMatrix
{
	int first_matrix_loop;
	int second_matrix_loop;
};

__host__ __device__ SubMatrix calculate_common_dimension(){
	int first_matrix_loop = 0;
	int second_matrix_loop = 0;
	// define a factor to minimize time
	if (TILEX > TILEY){
		first_matrix_loop = factor;
		second_matrix_loop = factor * TILEX / TILEY;
	}
	else{
		first_matrix_loop = factor * TILEY / TILEX;
		second_matrix_loop = factor;
	}
	struct SubMatrix sub_matrix = {first_matrix_loop, second_matrix_loop};
	return sub_matrix;
}

dim3 getDimGrid(const int m, const int n) {
	dim3 dimGrid(n/TILEX,n/TILEY);
	return dimGrid;
}
dim3 getDimBlock(const int m, const int n) {
	dim3 dimBlock(TILEX,TILEY);
	return dimBlock;
}
__global__ void kernelFunc(float* ad, float* bd, float* cd, const int m, const int n) {

	struct SubMatrix sub_matrix = calculate_common_dimension();

	__shared__ float A[TILEY][common_dim_of_matrixes];
	__shared__ float B[common_dim_of_matrixes][TILEX];

	int i = TILEY * by + ty;
	int j = TILEX * bx + tx;
	
	float C = 0;

	int number_of_iterations = common_dim_of_matrixes;
	int max_loop = max (sub_matrix.first_matrix_loop, sub_matrix.second_matrix_loop);
	for(int e = 0 ; e < n/number_of_iterations ; e++)
	{	
		if ( max_loop > sub_matrix.first_matrix_loop)
		{
			for(int k = 0; k < sub_matrix.first_matrix_loop ; k++){
				A[ty][k * TILEX + tx] = ad[i*n + (e*number_of_iterations + k * TILEX + tx)];
				B[k * TILEY + ty][tx] = bd[(e*number_of_iterations + k * TILEY + ty)*n + j];
			}
			for (int k = sub_matrix.first_matrix_loop; k < sub_matrix.second_matrix_loop; k++){
				B[k * TILEY + ty][tx] = bd[(e*number_of_iterations + k * TILEY + ty)*n + j];
			}
		}
		else if (max_loop > sub_matrix.second_matrix_loop){
			for(int k = 0; k < sub_matrix.second_matrix_loop ; k++){
				A[ty][k * TILEX + tx] = ad[i*n + (e*number_of_iterations + k * TILEX + tx)];
				B[k * TILEY + ty][tx] = bd[(e*number_of_iterations + k * TILEY + ty)*n + j];
			}
			for (int k = sub_matrix.second_matrix_loop; k < sub_matrix.first_matrix_loop; k++){
				A[ty][k * TILEX + tx] = ad[i*n + (e*number_of_iterations + k * TILEX + tx)];
			}

		}
		else{
			for(int k = 0; k < sub_matrix.second_matrix_loop ; k++){
				A[ty][k * TILEX + tx] = ad[i*n + (e*number_of_iterations + k * TILEX + tx)];
				B[k * TILEY + ty][tx] = bd[(e*number_of_iterations + k * TILEY + ty)*n + j];
			}
		}
		__syncthreads();

		for(int k=0; k < number_of_iterations;k++)
		{
			C += A[ty][k] * B[k][tx];
		}
		__syncthreads();
	}

	cd[i*n + j] = C;
}