#include "hip/hip_runtime.h"
// ONLY MODIFY THIS FILE

#include "scan2.h"
#include "gpuerrors.h"
#include "stdio.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// you may define other parameters here!
const int max_array_size = 1024;
const int max_scan_len_one_step = 1<<25;
// you may define other macros here!
#define max(X, Y) (((X) > (Y)) ? (X) : (Y))
// you may define other functions here!

__global__ void Blelloch(float* input, float* total_sum, int n)
{
    __shared__ float scan[max_array_size];

    scan[tx] = input[bx * n + tx];
    __syncthreads();

    int step;
    for(step = 2; step <= n; step = step* 2)
    {
        if(tx % step == step-1)
        {
            scan[tx] = scan[tx - (step/2)] + scan[tx];
        }
        __syncthreads();
    }

    if(tx == 0)
    {
        total_sum[bx] = scan[(n - 1)];
        scan[(n - 1)] = 0;
    }
    __syncthreads();


    //////////////////////////
    //  left    right 
    //    \      /
    //     \    /
    //      \  /
    //       \/
    //       /\    
    //      /  \   
    //     /    \  
    //    /      \  
    // right   left+right
    //////////////////////////
    
    float right;
    float left;
    for(int step = n; step >= 2; step /= 2)
    {
        if(tx % step == step-1)
        {
            left = scan[tx - (step/2)];
            right = scan[tx];
            scan[tx - (step/2)] = right;
            right += left; 
            scan[tx] = right;
        }
        __syncthreads();
    }

    input[bx * n + tx] = scan[tx];
    __syncthreads();
}

__global__ void add_sum_of_last_block(float* input, float* total_sum_prev_block, int n)
{
    input[bx * n + tx] += total_sum_prev_block[bx];
    __syncthreads();
}

void gpuKernel(float* a, float* c,int n) {
	
	int number_of_iteration = max(n/max_scan_len_one_step, 1);

	float* elemets_array;
	float* sum_of_blockes_array;
	float* scan_in_sum_of_blocks;
	float* total_sum;

	hipMalloc((void**)&elemets_array, n * sizeof(float));
	hipMalloc((void**)&sum_of_blockes_array, (n / max_array_size ) * sizeof(float));
	hipMalloc((void**)&scan_in_sum_of_blocks, (n / (max_array_size*max_array_size)) * sizeof(float));
	hipMalloc((void**)&total_sum, sizeof(float));

	int array_size = n;	
	
	for (int i = 0; i < number_of_iteration; i++){

		n = array_size / number_of_iteration;
		if (i == 0) {
			hipMemcpy(elemets_array, a , n * sizeof(float), hipMemcpyHostToDevice);
		}
		else {
			hipMemcpy(elemets_array, a+i*n-1 , n * sizeof(float), hipMemcpyHostToDevice);
		}
		Blelloch<<< n/max_array_size , max_array_size >>>(elemets_array,sum_of_blockes_array,max_array_size);
		Blelloch<<< n/(max_array_size*max_array_size) , max_array_size >>>(sum_of_blockes_array,scan_in_sum_of_blocks,max_array_size);
		Blelloch<<< 1 , n/(max_array_size*max_array_size) >>>(scan_in_sum_of_blocks,total_sum,n/(max_array_size*max_array_size));

		add_sum_of_last_block <<< n/(max_array_size*max_array_size) , max_array_size>>> (sum_of_blockes_array,scan_in_sum_of_blocks,max_array_size);
		add_sum_of_last_block <<< n/max_array_size , max_array_size>>> (elemets_array,sum_of_blockes_array,max_array_size);

		
		if (i == 0) {
			hipMemcpy(c, elemets_array+1, (n-1) * sizeof(float), hipMemcpyDeviceToHost);
		}
		else {
			hipMemcpy(c + i*n, elemets_array+2, (n-2) * sizeof(float), hipMemcpyDeviceToHost);
		}
		

		if (i == 0){
			c[(1+i)*n-1] = c[(1+i)*n-2] + a[(1+i)*n-1];
			a[(1+i)*n-1] = c[(1+i)*n-1];
		}
		else if (i == number_of_iteration -1){
			c[(1+i)*n-2] = c[(1+i)*n-3] + a[(1+i)*n-2];
			c[(1+i)*n-1] = c[(1+i)*n-2] + a[(1+i)*n-1];
		}
		else{
			c[(1+i)*n-2] = c[(1+i)*n-3] + a[(1+i)*n-2];
			c[(1+i)*n-1] = c[(1+i)*n-2] + a[(1+i)*n-1];
			a[(1+i)*n-1] = c[(1+i)*n-1];
		}

	}

	hipFree(elemets_array);
	hipFree(sum_of_blockes_array);
	hipFree(scan_in_sum_of_blocks);
	hipFree(total_sum);

}
	