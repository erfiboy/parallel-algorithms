#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE

#include "mm.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// you may change the following values
#define TILEX 16
#define TILEY 64

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

dim3 getDimGrid(const int m, const int n) {
       dim3 dimGrid(n/TILEX,n/TILEY);
       return dimGrid;
}
dim3 getDimBlock(const int m, const int n) {
       dim3 dimBlock(TILEX,TILEY);
       return dimBlock;
}

//-----------------------------------------------------------------------------
__global__ void kernelFunc(float* ad, float* bd, float* cd, const int m,const int n) {
       // write your GPU kernel function here
       // note that maximum # of threads per block is 1024
    int j = by*blockDim.y+ty;
    int i = bx*blockDim.x+tx;

	cd[j * n + i] = 0;

	for (int k = 0; k < n; k++) {
		cd[j * n + i] += ad[j * n + k] * bd[k * n + i];
	}

}